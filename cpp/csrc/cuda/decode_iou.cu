#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

 #include "decode_iou.h"
 #include "utils.h"
 #include <cstdio>
 
 #include <algorithm>
 #include <cstdint>
 
 #include <thrust/device_ptr.h>
 #include <thrust/sequence.h>
 #include <thrust/execution_policy.h>
 #include <thrust/gather.h>
 #include <thrust/tabulate.h>
 #include <thrust/count.h>
 #include <thrust/find.h>
#include <thrust/system/cuda/detail/cub/device/device_radix_sort.cuh>
#include <thrust/system/cuda/detail/cub/iterator/counting_input_iterator.cuh>
 
 namespace ryolo
 {
     namespace cuda
     {
 
         __global__ void softmax_kernel(const float *data, float *scores, float *conf, float *boxes, int num_elem)
         {
             int idx = threadIdx.x + blockIdx.x * blockDim.x;
             if (idx >= num_elem)
                 return;
             
 
             for (int k = 0; k < 5; ++k)
             { // 5 == num_anchor_per_point
 
             
                 boxes[idx + num_elem * (6 * k + 0)] = 1 / (1.0f + expf(data[idx + num_elem * (8 * k + 0)] * -1.0));
                 boxes[idx + num_elem * (6 * k + 1)] = 1 / (1.0f + expf(data[idx + num_elem * (8 * k + 1)] * -1.0));
                 boxes[idx + num_elem * (6 * k + 2)] = 1 / (1.0f + expf(data[idx + num_elem * (8 * k + 2)] * -1.0));
                 boxes[idx + num_elem * (6 * k + 3)] = 1 / (1.0f + expf(data[idx + num_elem * (8 * k + 3)] * -1.0));
                 boxes[idx + num_elem * (6 * k + 4)] = 1 / (1.0f + expf(data[idx + num_elem * (8 * k + 4)] * -1.0));
                 float temp = expf(2 * data[idx + num_elem * (8 * k + 5)]);
                 boxes[idx + num_elem * (6 * k + 5)] = (temp - 1) / (temp + 1);
                 
                 // for(int i =0; i<6; i++)
                 // {
                 //     printf("%f  ",boxes[idx + num_elem * (6 * k + i)]);
                 // }
                 // printf("\n");
                 float score = data[idx + num_elem * (8 * k + 6)];
                 // printf("\n%f", score);
                 score = 1 / (1.0f + expf(score * -1.0));
 
                 scores[idx + num_elem * 1 * k] = score;
                 float cls = data[idx + num_elem * (8 * k + 7)];
                 cls = 1 / (1.0f + expf(cls * -1.0));
                 cls = cls * score;
                 conf[idx + num_elem * 1 * k] = cls;
             }
         }
 
         int decode(int batch_size,
                    const void *const *inputs, void *const *outputs,
                    size_t num_anchors, const std::vector<float> &anchors,
                    int top_n, size_t f_size, float score_thresh,
                    int stride,
                    void *workspace, size_t workspace_size, hipStream_t stream)
         {
 
             if (!workspace || !workspace_size)
             {
                 // scratch space size cub style
                 workspace_size = get_size_aligned<float>(anchors.size());   // anchors
                 workspace_size += get_size_aligned<bool>(num_anchors);      // flags
                 workspace_size += get_size_aligned<int>(num_anchors);       // indices
                 workspace_size += get_size_aligned<int>(num_anchors);       // indices_sorted
                 workspace_size += get_size_aligned<float>(num_anchors);     // scores
                 workspace_size += get_size_aligned<float>(num_anchors);     // scores_sorted
                 workspace_size += get_size_aligned<float>(num_anchors);     // scores_softmax
                 workspace_size += get_size_aligned<float>(num_anchors);     // conf
                 workspace_size += get_size_aligned<float>(num_anchors * 6); // in_boxes
                workspace_size += get_size_aligned<float>(num_anchors);     // classes
 
                 size_t temp_size_flag = 0;
                 thrust::cuda_cub::hipcub::DeviceSelect::Flagged((void *)nullptr, temp_size_flag,
                                            thrust::cuda_cub::hipcub::CountingInputIterator<int>(num_anchors),
                                            (bool *)nullptr, (int *)nullptr, (int *)nullptr, num_anchors);
                 size_t temp_size_sort = 0;
                 thrust::cuda_cub::hipcub::DeviceRadixSort::SortPairsDescending((void *)nullptr, temp_size_sort,
                                                           (float *)nullptr, (float *)nullptr, (int *)nullptr, (int *)nullptr, num_anchors);
                 workspace_size += std::max(temp_size_flag, temp_size_sort);
 
                 return workspace_size;
             }
 
             auto anchors_d = get_next_ptr<float>(anchors.size(), workspace, workspace_size);
             hipMemcpyAsync(anchors_d, anchors.data(), anchors.size() * sizeof *anchors_d, hipMemcpyHostToDevice, stream);
 
             auto on_stream = thrust::cuda::par.on(stream);
 
             auto flags = get_next_ptr<bool>(num_anchors, workspace, workspace_size);  // used for filtering flags by threshold
             auto indices = get_next_ptr<int>(num_anchors, workspace, workspace_size); // used for filtering index by threshold
             auto indices_sorted = get_next_ptr<int>(num_anchors, workspace, workspace_size);
             auto scores = get_next_ptr<float>(num_anchors, workspace, workspace_size);
             auto scores_sorted = get_next_ptr<float>(num_anchors, workspace, workspace_size);
             auto scores_softmax = get_next_ptr<float>(num_anchors, workspace, workspace_size);
             auto conf = get_next_ptr<float>(num_anchors, workspace, workspace_size);
 
             auto in_boxes = get_next_ptr<float>(num_anchors * 6, workspace, workspace_size);
 
             int thread_count;
             int num_anchor = 5;
 
             for (int batch = 0; batch < batch_size; batch++)
             {
                 auto in_data = static_cast<const float *>(inputs[0]) + batch * num_anchors * 8; // cx,cy,w,h,cos,sin,score,cls
                 auto out_scores = static_cast<float *>(outputs[0]) + batch * top_n;
                 auto out_boxes = static_cast<float6 *>(outputs[1]) + batch * top_n;
                 auto out_classes = static_cast<float *>(outputs[2]) + batch * top_n;
 
                 // sigmoid activation
                 const int thread_count_ = 128;
                 int num_elem = f_size * f_size;
                 thread_count = (num_elem < thread_count_) ? num_elem : thread_count_;
                 softmax_kernel<<<(num_elem + thread_count - 1) / thread_count, thread_count, 0, stream>>>(in_data, scores_softmax, conf, in_boxes, num_elem);
 
                 // Discard scores below threshold
                 thrust::transform(on_stream, scores_softmax, scores_softmax + num_anchors, flags, thrust::placeholders::_1 > score_thresh);
 
                 int *num_selected = reinterpret_cast<int *>(indices_sorted);
                 thrust::cuda_cub::hipcub::DeviceSelect::Flagged(workspace, workspace_size,
                                            thrust::cuda_cub::hipcub::CountingInputIterator<int>(0),
                                            flags, indices, num_selected, num_anchors, stream);
                 hipStreamSynchronize(stream);
                 int num_detections = *thrust::device_pointer_cast(num_selected);
 
                 // Only keep top n scores
                 auto indices_filtered = indices;
                 if (num_detections > top_n)
                 {
                     // lấy score theo indices đã chọn ở trên, sort index theo score, đẩy vào scores
                     thrust::gather(on_stream, indices, indices + num_detections, scores_softmax, scores);
                     // sort các giá trị trong scores đẩy vào scores_sorted để lấy n giá trị
                     thrust::cuda_cub::hipcub::DeviceRadixSort::SortPairsDescending(workspace, workspace_size,
                                                               scores, scores_sorted, indices, indices_sorted, num_detections, 0, sizeof(*scores) * 8, stream);
                     indices_filtered = indices_sorted;
                     num_detections = top_n;
                 }
 
                 // Gather boxes
                 // bool has_anchors = !anchors.empty();
                 thrust::transform(on_stream, indices_filtered, indices_filtered + num_detections,
                                   thrust::make_zip_iterator(thrust::make_tuple(out_scores, out_boxes, out_classes)),
                                   [=] __device__(int i)
                                   {
                                       int x = i % f_size;
                                       int y = (i / f_size) % f_size;
                                       int a = (i / f_size / f_size) % num_anchor;
 
                                       float cx = in_boxes[((a * 6 + 0) * f_size + y) * f_size + x];
                                       float cy = in_boxes[((a * 6 + 1) * f_size + y) * f_size + x];
                                       float w = in_boxes[((a * 6 + 2) * f_size + y) * f_size + x];
                                       float h = in_boxes[((a * 6 + 3) * f_size + y) * f_size + x];
                                       float cos = in_boxes[((a * 6 + 4) * f_size + y) * f_size + x];
                                       float sin = in_boxes[((a * 6 + 5) * f_size + y) * f_size + x];
 
                                       cx = (cx * 2.0f - 0.5f + x) * stride;
                                       cy = (cy * 2.0f - 0.5f + y) * stride;
                                       w = w * w * 4 * anchors_d[a * 2 + 0];
                                       h = h * h * 4 * anchors_d[a * 2 + 1];                                       
                                       
                                     //   convert to xyxyxyxy
                                       float x0 = -w / 2.0f;
                                       float x1 = w / 2.0f;
                                       float y0 = -h / 2.0f;
                                       float y1 = h / 2.0f;
 
                                       float xyxyxyxy[4][2] = {{x0, y0}, {x1, y0}, {x1, y1}, {x0, y1}};
                                       float R[2][2] = {{cos, sin}, {sin, cos}};
                                       float temp[4][2] = {{0, 0}, {0, 0}, {0, 0}, {0, 0}};
                                       for (int m = 0; m < 4; ++m)
                                           for (int j = 0; j < 2; ++j)
                                           {
                                               for (int k = 0; k < 2; ++k)
                                               {
                                                   temp[m][j] += xyxyxyxy[m][k] * R[k][j];
                                               }
                                           }
                                       for (int m = 0; m < 4; ++m)
                                       {
                                           temp[m][0] += cx;
                                           temp[m][1] += cy;
                                       }
                                       float x_min = temp[0][0];
                                       float x_max = temp[0][0];
                                       for(int m=1; m<4; m++)
                                       {
                                         if (temp[m][0]<x_min)
                                         x_min = temp[m][0];
                                         if (temp[m][0]>x_max)
                                         x_max = temp[m][0];
                                       }
                                       float y_min = temp[0][1];
                                       float y_max = temp[0][1];
                                       for(int m=1; m<4; m++)
                                       {
                                         if (temp[m][1]<y_min)
                                         y_min = temp[m][1];
                                         if (temp[m][1]>y_max)
                                         y_max = temp[m][1];
                                       }
                                       float6 box = make_float6(make_float4(x_min, y_min, x_max,y_max),make_float2(sin, cos));
                               
                                       return thrust::make_tuple(conf[i], box, 0);
                                   });
 
                 // Zero-out unused scores
                 if (num_detections < top_n)
                 {
                     thrust::fill(on_stream, out_scores + num_detections,
                                  out_scores + top_n, 0.0f);
                     thrust::fill(on_stream, out_classes + num_detections,
                                 out_classes + top_n, 0.0f);
                 }
             }
 
             return 0;
         }
 
     }
 }
